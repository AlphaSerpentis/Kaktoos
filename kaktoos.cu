// IDE indexing
#ifdef __JETBRAINS_IDE__
#define __host__
#define __device__
#define __shared__
#define __constant__
#define __global__
#define __CUDACC__
#include <device_functions.h>
#include <__clang_cuda_builtin_vars.h>
#include <__clang_cuda_intrinsics.h>
#include <__clang_cuda_math_forward_declares.h>
#include <__clang_cuda_complex_builtins.h>
#include <__clang_cuda_cmath.h>
#endif


#include <hip/hip_runtime.h>
#include <stdint.h>
#include <memory.h>
#include <stdio.h>
#include <time.h>

#define RANDOM_MULTIPLIER_LONG 0x5DEECE66DULL

#define RANDOM_MULTIPLIER RANDOM_MULTIPLIER_LONG
#define RANDOM_ADDEND 0xBULL
#define RANDOM_MASK ((1ULL << 48) - 1)
#define RANDOM_SCALE 1

#define FAST_NEXT_INT

// Random::next(bits)
__device__ inline uint32_t random_next(uint64_t *random, int32_t bits) {
    *random = (*random * RANDOM_MULTIPLIER + RANDOM_ADDEND) & RANDOM_MASK;
    return (uint32_t)(*random >> (48 - bits));
}

// new Random(seed)
#define get_random(seed) ((Random)((seed ^ RANDOM_MULTIPLIER_LONG) & RANDOM_MASK))
#define get_random_unseeded(state) ((Random) ((state) * RANDOM_SCALE))

__device__ int32_t next_int_unknown(uint64_t *seed, int16_t bound) {
    if ((bound & -bound) == bound) {
        *seed = (*seed * RANDOM_MULTIPLIER + RANDOM_ADDEND) & RANDOM_MASK;
        return (int32_t)((bound * (*seed >> 17)) >> 31);
    }

    int32_t bits, value;
    do {
        *seed = (*seed * RANDOM_MULTIPLIER + RANDOM_ADDEND) & RANDOM_MASK;
        bits = *seed >> 17;
        value = bits % bound;
    } while (bits - value + (bound - 1) < 0);
    return value;
}

// Random::nextInt(bound)
__device__ inline uint32_t random_next_int(uint64_t *random) {
    return random_next(random, 31) % 3;
}

#define TOTAL_WORK_SIZE (1LL << 48)

#define WORK_UNIT_SIZE (1LL << 20)
#define BLOCK_SIZE 256

__device__ inline int8_t extract(int32_t heightMap[], int32_t i) {
    return (int8_t)(heightMap[(i) >> 2] >> ((i & 0b11) << 3) & 0xFF);
}

__device__ inline void increase(int32_t heightMap[], int32_t i) {
    heightMap[i >> 2] += 1 << ((i & 0b11) << 3);
}

__global__ void crack(uint64_t seed_offset, int32_t *num_seeds, uint64_t *seeds) {
    uint64_t originalSeed = blockIdx.x * blockDim.x + threadIdx.x + seed_offset;
    if (originalSeed >= TOTAL_WORK_SIZE)
        return;
    uint64_t seed = originalSeed;

    int16_t wantedCactusHeight = 8;
    int8_t floorLevel = 63;
    int16_t attemptsCount = 10;
    int32_t heightMap[256];

    for (int32_t temp = 0; temp < 256; temp++) {
        heightMap[temp] = floorLevel | floorLevel << 8 | floorLevel << 16 | floorLevel << 24;
    }

    int16_t currentHighestPos = 0;
    int16_t terrainHeight;
    int16_t initialPosX, initialPosY, initialPosZ;
    int16_t posX, posY, posZ;
    int16_t offset, posMap;

    int16_t i, a, j;

    for (i = 0; i < attemptsCount; i++) {
        // Keep, most threads finish early this way
        if (wantedCactusHeight - extract(heightMap, currentHighestPos) + floorLevel > 9 * (attemptsCount - i))
            return;

        initialPosX = random_next(&seed, 4) + 8;
        initialPosZ = random_next(&seed, 4) + 8;
        terrainHeight = (extract(heightMap, initialPosX + initialPosZ * 32) + 1) * 2;

        initialPosY = next_int_unknown(&seed, terrainHeight);

        for (a = 0; a < 10; a++) {
            posX = initialPosX + random_next(&seed, 3) - random_next(&seed, 3);
            posY = initialPosY + random_next(&seed, 2) - random_next(&seed, 2);
            posZ = initialPosZ + random_next(&seed, 3) - random_next(&seed, 3);

            posMap = posX + posZ * 32;
            // Keep
            if (posY <= extract(heightMap, posMap) && posY >= 0)
                continue;

            offset = 1 + next_int_unknown(&seed, random_next_int(&seed) + 1);

            for (j = 0; j < offset; j++) {
                if ((posY + j - 1) > extract(heightMap, posMap) || posY < 0) continue;
                if ((posY + j) <= extract(heightMap, (posX + 1) + posZ * 32) && posY >= 0) continue;
                if ((posY + j) <= extract(heightMap, posX + (posZ - 1) * 32) && posY >= 0) continue;
                if ((posY + j) <= extract(heightMap, (posX - 1) + posZ * 32) && posY >= 0) continue;
                if ((posY + j) <= extract(heightMap, posX + (posZ + 1) * 32) && posY >= 0) continue;

                increase(heightMap, posMap);

                if (extract(heightMap, currentHighestPos) < extract(heightMap, posMap)) {
                    currentHighestPos = posMap;
                }
            }
        }

        if (extract(heightMap, currentHighestPos) - floorLevel >= wantedCactusHeight) {
            int32_t index = atomicAdd(num_seeds, 1);
            seeds[index] = originalSeed;
            return;
        }
    }
}

#define GPU_COUNT 1

struct GPU_Node {
    int GPU;
    int* num_seeds;
    uint64_t* seeds;
};

void setup_gpu_node(GPU_Node* node, int32_t gpu) {
    hipSetDevice(gpu);
    node->GPU = gpu;
    hipMallocManaged(&node->num_seeds, sizeof(*node->num_seeds));
    hipMallocManaged(&node->seeds, (1LL << 10)); // approx 1kb
}


GPU_Node nodes[GPU_COUNT];
int main() {
    printf("Searching %lld total seeds...\n", TOTAL_WORK_SIZE);

    FILE* out_file = fopen("kaktoos_seeds.txt", "w");

    for(int32_t i = 0; i < GPU_COUNT; i++) {
        setup_gpu_node(&nodes[i],i);
    }


    uint64_t count = 0;
    time_t startTime = time(NULL), currentTime;
    for (uint64_t offset = 0; offset < TOTAL_WORK_SIZE;) {

        for(int32_t gpu_index = 0; gpu_index < GPU_COUNT; gpu_index++) {
            hipSetDevice(gpu_index);
            *nodes[gpu_index].num_seeds = 0;
            crack<<<WORK_UNIT_SIZE / BLOCK_SIZE, BLOCK_SIZE>>> (offset, nodes[gpu_index].num_seeds, nodes[gpu_index].seeds);
            offset += WORK_UNIT_SIZE;
        }

        for(int32_t gpu_index = 0; gpu_index < GPU_COUNT; gpu_index++) {
            hipSetDevice(gpu_index);
            hipDeviceSynchronize();

            for (int32_t i = 0, e = *nodes[gpu_index].num_seeds; i < e; i++) {
                fprintf(out_file, "%lld\n", (long long int)nodes[gpu_index].seeds[i]);
            }
            fflush(out_file);
            count += *nodes[gpu_index].num_seeds;
        }

        time(&currentTime);
        int timeElapsed = (int)(currentTime - startTime);
        uint64_t numSearched = offset + WORK_UNIT_SIZE;
        double speed = (double)numSearched / (double)timeElapsed / 1000000.0;
        printf("Searched %lld seeds, found %lld matches . Time elapsed: %ds. Speed: %.2fm seeds/s.\n", (long long int)numSearched, (long long int)count, timeElapsed, speed);
    }

    fclose(out_file);

}
